#include "cub_wrapper.cuh"



size_t cub_prefixSum(size_t *d_in, size_t *d_out, size_t ns, void *d_temp, size_t t_size){

    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


size_t last_nb = 0;
size_t tot = 0;
void *dtmp = NULL;
size_t tsb = 0;

//void     *d_temp_storage = NULL;
//size_t   temp_storage_bytes = 0;
hipcub::DeviceScan::ExclusiveSum(dtmp, tsb, d_in, d_out, ns);
// Allocate temporary storage
hipMalloc(&dtmp, tsb);
printf("tsb= %zu\n",tsb);
hipcub::DeviceScan::ExclusiveSum(dtmp, tsb, d_in, d_out, ns);

// hipcub::DeviceScan::ExclusiveSum(d_temp, t_size, d_in, d_out, ns);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cub prefixsum time=%.6f\n", milliseconds/1000.0 );



return last_nb + tot;
}
