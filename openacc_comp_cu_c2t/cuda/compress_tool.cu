#include "hip/hip_runtime.h"
#include "compress_tool.cuh"


// const size_t grid_size2 = reqBytesLen * len/block_size;

typedef union ldouble
{
    double value;
    unsigned long lvalue;
    unsigned char byte[8];
} ldouble;

__device__ inline short getPrecisionReqLength_double(double precision)
{
	ldouble lbuf;
	lbuf.value = precision;
	long lvalue = lbuf.lvalue;
	
	int expValue = (int)((lvalue & 0x7FF0000000000000) >> 52);
	expValue -= 1023;

	return (short)expValue;
}

__device__ inline short computeReqLength_double_MSST19(double realPrecision)
{
	short reqExpo = getPrecisionReqLength_double(realPrecision);
	return 12-reqExpo;
}

__global__ void set_value(){
    size_t K = 1024;
    size_t GB_num = 1;
    size_t byte_num = 8;
    len = GB_num*K/byte_num*K*K;
    chk_size = 32;
    num_chk = len/ chk_size;
    pwrpr = 1E-5;
    t_size = 699903;
    reqLen =  computeReqLength_double_MSST19(pwrpr);
    reqBytesLen = reqLen/8;
	resiBitsLen = reqLen%8;	
    totLeadNums = chk_size;
    if(totLeadNums%4==0)
        totLeadNums = totLeadNums*2/byte_num;
	else
        totLeadNums = totLeadNums*2/byte_num+1;

    totLeadNums *= num_chk;
    // totBit2ByteNum = len;
    totBit2ByteNum = ((resiBitsLen * (chk_size))/byte_num + 1) * (num_chk);
    totMidByteNum = reqBytesLen * (len);
    compBufSize = totLeadNums + totBit2ByteNum + totMidByteNum;
    midByteOffsetPacked = totLeadNums + totBit2ByteNum;
    midByteOffset = 0;
    block_size = 128;
}
void initialize_value(size_t* h_len,size_t* h_chk_size, size_t* h_num_chk, double* h_pwrpr, size_t* h_t_size, short* h_reqLen,\
    int* h_reqBytesLen, int* h_resiBitsLen, size_t* h_totLeadNums, size_t* h_totBit2ByteNum, size_t *h_totMidByteNum,\
    size_t* h_compBufSize,size_t* h_midByteOffset, size_t* h_midByteOffsetPacked,size_t* h_block_size){
    
    set_value<<<1,1>>>();

    // *h_chk_size = 32;
    hipMemcpyFromSymbol(h_len, HIP_SYMBOL(len), sizeof(size_t),0, hipMemcpyDeviceToHost);
    // *h_len = len;
    hipMemcpyFromSymbol(h_chk_size, HIP_SYMBOL(chk_size), sizeof(size_t),0, hipMemcpyDeviceToHost);
    // *h_chk_size = chk_size;
    // hipDeviceSynchronize();
    
    hipMemcpyFromSymbol(h_num_chk, HIP_SYMBOL(num_chk), sizeof(size_t),0, hipMemcpyDeviceToHost);
    // printf("len = %zu\n",*h_len);

    // *h_num_chk = num_chk;
    hipMemcpyFromSymbol(h_pwrpr, HIP_SYMBOL(pwrpr), sizeof(double),0, hipMemcpyDeviceToHost);
    // *h_pwrpr = pwrpr;
    hipMemcpyFromSymbol(h_t_size, HIP_SYMBOL(t_size), sizeof(size_t),0, hipMemcpyDeviceToHost);
    // *h_t_size = t_size;
    
    
    // hipMemcpy(h_reqLen, &reqLen, sizeof(short), hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(h_reqLen, HIP_SYMBOL(reqLen), sizeof(short),0, hipMemcpyDeviceToHost);

    // *h_reqLen = reqLen;
    
    hipMemcpyFromSymbol(h_reqBytesLen, HIP_SYMBOL(reqBytesLen), sizeof(int),0, hipMemcpyDeviceToHost);  
    // *h_reqBytesLen = reqBytesLen;
    hipMemcpyFromSymbol(h_resiBitsLen, HIP_SYMBOL(resiBitsLen), sizeof(int),0, hipMemcpyDeviceToHost);  
    // *h_resiBitsLen = resiBitsLen;
    hipMemcpyFromSymbol(h_totLeadNums, HIP_SYMBOL(totLeadNums), sizeof(size_t),0, hipMemcpyDeviceToHost);  
    // *h_totLeadNums = totLeadNums;
    hipMemcpyFromSymbol(h_totBit2ByteNum, HIP_SYMBOL(totBit2ByteNum), sizeof(size_t),0, hipMemcpyDeviceToHost); 
    // *h_totBit2ByteNum = totBit2ByteNum;
    hipMemcpyFromSymbol(h_totMidByteNum, HIP_SYMBOL(totMidByteNum), sizeof(size_t),0, hipMemcpyDeviceToHost);  
    // *h_totMidByteNum = totMidByteNum;
    hipMemcpyFromSymbol(h_compBufSize, HIP_SYMBOL(compBufSize), sizeof(size_t),0, hipMemcpyDeviceToHost);  
    // *h_compBufSize = compBufSize;
    hipMemcpyFromSymbol(h_midByteOffset, HIP_SYMBOL(midByteOffset), sizeof(size_t),0, hipMemcpyDeviceToHost); 
    // *h_midByteOffset = midByteOffset;
    hipMemcpyFromSymbol(h_midByteOffsetPacked, HIP_SYMBOL(midByteOffsetPacked), sizeof(size_t),0, hipMemcpyDeviceToHost); 
    // *h_midByteOffsetPacked = midByteOffsetPacked; 
    hipMemcpyFromSymbol(h_block_size, HIP_SYMBOL(block_size), sizeof(size_t),0, hipMemcpyDeviceToHost); 
    // *h_block_size = block_size;
}


void HostMemAlloc(void ** pHost, size_t size){
    hipHostAlloc(pHost, size, hipHostMallocDefault);
}

void DeviceMemAlloc(void ** dHost, size_t size){
    hipError_t a = hipMalloc(dHost, size);
    if (a!=hipSuccess){
        printf("Error happens with device memory malloc\n");
        printf("%s\n",hipGetErrorString(a));
        exit(1);
    }
}

void HostMemFree(void * pHost){
    hipHostFree(pHost); 
}

void DeviceMemFree(void * dHost){
    hipFree(dHost); 
}

float Host2Device(void* dst, const void* src, size_t count){
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipMemcpy(dst,src,count,hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    return milliseconds/1000.0;
}

float Device2Host(void* dst, const void* src, size_t count){
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipMemcpy(dst,src,count,hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    return milliseconds/1000.0;
}

__global__ void comp_p1_kernal(double* uncompBuffer, unsigned char* compBuffer, size_t* prefixSum, size_t* prefSums, unsigned char* midBytes4Packing){
    

    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < num_chk){
    size_t strtChkMidBytes = midByteOffset+i*reqBytesLen*len/num_chk;
    size_t strtLdNum = i*totLeadNums/num_chk;
    size_t strtResibits = i*totBit2ByteNum/num_chk + totLeadNums;
    size_t packedNum = 0;
    unsigned char mbNbs[4]; // shared memo or registers?
    for(size_t j=0;j<chk_size;j+=4) // sequential
    {
         for (int k=0;k<4;k++)
         {
            if(i*chk_size+j+k == len-1)
            {
                last_number = uncompBuffer[i*chk_size+j+k];
                break;
            }
             double prevVal = (j+k)>0?uncompBuffer[i*chk_size+j+k-1]:0.;
            // double prevVal = (j+k)>0?uncompBuffer[i*chk_size]:0.;
             double currVal = uncompBuffer[i*chk_size+j+k];
             char *ptr0 = NULL, *ptr1 = NULL; 
             int ldNum = 0;
             ptr0 = (char *)&prevVal;
             ptr1 = (char *)&currVal;
             
             /* record number of mid-bytes different from previous */
             if((j+k)!=0)
             {
                for(int l=0;l<reqBytesLen;l++)
                {
                    if (*(ptr0+7-l) == *(ptr1+7-l))
                       ldNum++;
                    else
                        break;
                }
             }
             mbNbs[k] = reqBytesLen - ldNum;
             prefixSum[i] += reqBytesLen - ldNum;

             /* record mid-bytes different from previous */
             for(int l=ldNum;l<reqBytesLen;l++) 
             {
                 //midBytes4Packing[strtChkMidBytes] = *(ptr1+7-l);    
                 midBytes4Packing[strtChkMidBytes] = *(ptr1+7-l);    
                 strtChkMidBytes++;
             }

             /* record and pack residue bits */
            //  unsigned char resiBits = *(ptr1+7-reqBytesLen);
            //  resiBits = (resiBits >> (8-resiBitsLen)) << (8-resiBitsLen);
             
            //  compBuffer[strtResibits+j + k] =  resiBits;
            unsigned char resiBits = *(ptr1+7-reqBytesLen);
             resiBits = (resiBits >> (8-resiBitsLen)) << (8-resiBitsLen);
             size_t posByBits = packedNum * resiBitsLen;
             size_t headByteId = posByBits/8;
             size_t tailByteId = (posByBits+resiBitsLen)/8;
             int headBits = 8-posByBits%8;  // 
             headBits = headBits > resiBitsLen? resiBitsLen: headBits;

             compBuffer[strtResibits+headByteId] = compBuffer[strtResibits+headByteId] | (resiBits >> (posByBits%8));
             compBuffer[strtResibits+tailByteId] = compBuffer[strtResibits+tailByteId] | (resiBits << headBits);
             packedNum++;
         }

         /* pack numbers of mid-bytes */
         int tmp = 0;
         int ctr = 0;
         for(int k = 0;k<4;k++)
         {
            unsigned int type = mbNbs[k];
            switch(type)
            {
                case 0: 
                    break;
                case 1:
                    tmp = (tmp | (1 << (6-ctr*2)));
                    break;
                case 2:
                    tmp = (tmp | (2 << (6-ctr*2)));
                    break;
                case 3:
                    tmp = (tmp | (3 << (6-ctr*2)));
                    break;
                default:
                    ;
            }
            ctr++;
        }
        compBuffer[strtLdNum] = (unsigned char)tmp;    
        strtLdNum++;
    } // end intra-chunk (sequential) for-loop
    }
}

__global__ void comp_p2_kernal(unsigned char* compBuffer, size_t* prefixSum, size_t* prefSums, unsigned char* midBytes4Packing){
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i<totMidByteNum){
        size_t real_idx = i/chk_size/reqBytesLen;
	    size_t inchk_ofs = i%(chk_size*reqBytesLen);
	    if(inchk_ofs<prefixSum[real_idx])
	    { 
		    real_idx = prefSums[real_idx];
		    real_idx += inchk_ofs;
		    compBuffer[midByteOffsetPacked+real_idx] = midBytes4Packing[i];
	    }
    }
}

__global__ void comp_p2_kernal_v2(unsigned char* compBuffer, size_t* prefixSum, size_t* prefSums, unsigned char* midBytes4Packing){
    size_t z = blockIdx.x * blockDim.x + threadIdx.x;
    size_t local_chk_size = 2;
    size_t g = z * local_chk_size;
    if(g<totMidByteNum){
        for (size_t i = g;i<g+local_chk_size;i++){
            size_t real_idx = i/chk_size/reqBytesLen;
	        size_t inchk_ofs = i%(chk_size*reqBytesLen);
	        if(inchk_ofs<prefixSum[real_idx])
	        { 
		        real_idx = prefSums[real_idx];
		        real_idx += inchk_ofs;
		        compBuffer[midByteOffsetPacked+real_idx] = midBytes4Packing[i];
            
                // printf("%x ",compBuffer[midByteOffsetPacked+real_idx] );
            
	        }
        }
    }
}

float comp_p1(double* uncompBuffer, unsigned char* compBuffer, size_t* prefixSum, size_t* prefSums, \
    unsigned char* midBytes4Packing,const size_t grid_size, const size_t block_size){
    
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    printf("grid_size  = %zu\n",grid_size);
    comp_p1_kernal<<<grid_size,block_size>>>(uncompBuffer,compBuffer,prefixSum,prefSums,midBytes4Packing);
    
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    return milliseconds/1000.0;
}

float comp_p2(unsigned char* compBuffer, size_t* prefixSum, size_t* prefSums, unsigned char* midBytes4Packing,\
    const size_t grid_size, const size_t block_size){
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    comp_p2_kernal<<<grid_size,block_size>>>(compBuffer,prefixSum,prefSums,midBytes4Packing);

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    return milliseconds/1000.0;
}

__global__ void decomp_p1_kernal(unsigned char* compBuffer, size_t* prefixSum, double* local_prefSum){
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    
    if(i < num_chk){
        size_t strtLdNum = i*totLeadNums/num_chk;
        
        
        size_t result_preSum = 0;
        for(size_t j=0;j<chk_size;j+=4) // sequential
        {
            unsigned char tmp = compBuffer[strtLdNum];
            for (int k=0;k<4;k++)
            {
                size_t ldNum = 0;
                ldNum = (tmp>>(6-k*2)) & 0x0003;
                local_prefSum[i*chk_size + j + k] = result_preSum;  
                result_preSum += ldNum;              
            }
            strtLdNum++;
        }
        prefixSum[i] =  result_preSum; 

    }

}

__global__ void decomp_p1_kernal_v2(unsigned char* compBuffer, size_t* prefixSum, double* local_prefSum){
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    
    
    if(i < num_chk){
        size_t strtLdNum = i*totLeadNums/num_chk;
        
        
        size_t result_preSum = 0;
        for(size_t j=0;j<chk_size;j+=4) // sequential
        {
            unsigned char tmp = compBuffer[strtLdNum];
            for (int k=0;k<4;k++)
            {
                size_t ldNum = 0;
                ldNum = (tmp>>(6-k*2)) & 0x0003;
                local_prefSum[i*chk_size+j+k] = result_preSum;  
                result_preSum += ldNum;              
            }
            strtLdNum++;
        }
        prefixSum[i] =  result_preSum;
    }

}

__global__ void decomp_p2_kernal(double* uncompBuffer, unsigned char* compBuffer, size_t* prefSums){
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    
    if( (i < num_chk)){
        size_t strtResibits = i*totBit2ByteNum/num_chk + totLeadNums;
        size_t strMidBytes = midByteOffsetPacked; 
        double prev_num = 0.0;
        size_t packedNum = 0;
        // if(i == num_chk -1)
        //     prefSums[i+1] = prefSums[i] + prefixSum[i];
        for(size_t j=0;j<chk_size;j+=4) // sequential
        {
            for (int k=0;k<4;k++)
            {
                if(i*chk_size+j+k == len-1)
                {
                    uncompBuffer[i*chk_size+j+k] = last_number;
                    break;
                }
                size_t local_id = i * chk_size + j + k;
                int prev_len = 1;
                char resiBits = 0;
                size_t posByBits = packedNum * resiBitsLen;
                size_t headByteId = posByBits/8;
                size_t tailByteId = (posByBits+resiBitsLen)/8;
                int headBits = 8-posByBits%8;  // 
                int head_str_point = posByBits%8;
                headBits = headBits > resiBitsLen? resiBitsLen: headBits;
                int tailBits = resiBitsLen - headBits;
                char headPart = compBuffer[strtResibits+headByteId];
                headPart = headPart>>(8-headBits-head_str_point)<<(8-headBits);
                char tailPart = compBuffer[strtResibits+tailByteId]>> (8-tailBits)<<(8-tailBits-headBits);
                resiBits = resiBits | headPart;
                resiBits = resiBits | tailPart;
                packedNum++;
                int conc_len = ((j+k)==chk_size-1)?prefSums[i+1]-prefSums[i]- uncompBuffer[local_id]:uncompBuffer[local_id+1] - uncompBuffer[local_id];
                size_t conc_offset = prefSums[i] + uncompBuffer[local_id];
                int conc_same_bytes = reqBytesLen -conc_len ;
                char conc_value[8];
                char* ptr_z;
                ptr_z = (char*) &prev_num;
                for(int z = 0; z< 8;z++){
                    conc_value[z] = *(ptr_z+z);
                }
                for(int z = conc_same_bytes ; z< reqBytesLen;z++){
                    conc_value[7-z] = compBuffer[strMidBytes + conc_offset + z - conc_same_bytes];
                }
                conc_value[7-reqBytesLen] = resiBits;
                double* ptr = NULL;
                double tmp;
                ptr = (double*)conc_value;
                tmp = *ptr;
                prev_num = tmp; 
                
                uncompBuffer[i * chk_size+j+k] = tmp;
                
            }
        }
    }

}



float decomp_p1(unsigned char* compBuffer, size_t* prefixSum, double* local_prefSum,\
    const size_t grid_size, const size_t block_size){
        
        float milliseconds = 0;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        
        decomp_p1_kernal<<<grid_size,block_size>>>(compBuffer,prefixSum,local_prefSum);
    
        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);

        return milliseconds/1000.0;
}



float decomp_p2(double* uncompBuffer, unsigned char* compBuffer, size_t* prefSums,\
    const size_t grid_size, const size_t block_size){

       
        float milliseconds = 0;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        
        decomp_p2_kernal<<<grid_size,block_size>>>(uncompBuffer,compBuffer,prefSums);
        
        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        
        return milliseconds/1000.0;
}
void set_GPUID(int id){
    hipSetDevice ( id ) ;
}

